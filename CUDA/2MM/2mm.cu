#include "hip/hip_runtime.h"
#include <memory>

#include <noarr/structures_extended.hpp>
#include <noarr/structures/extra/traverser.hpp>
#include <noarr/structures/interop/bag.hpp>
#include <noarr/structures/interop/serialize_data.hpp>
#include <noarr/structures/interop/cuda_traverser.cuh>

#include "common.hpp"
#include "defines.cuh"
#include "2mm.cuh"

using num_t = DATA_TYPE;

namespace {

// initialize data
void init(num_t &alpha, num_t &beta, auto A, auto B, auto C, auto D) {
	// A: ni x nk
	// B: nk x nj
	// C: nl x nj
	// D: ni x nl
	alpha = 32412;
	beta = 2123;

	noarr::traverser(A).for_each([=](auto state) {
		auto [i, k] = noarr::get_indices<'i', 'k'>(state);
		A[state] = ((num_t) i * k) / NI;
	});

	noarr::traverser(B).for_each([=](auto state) {
		auto [k, j] = noarr::get_indices<'k', 'j'>(state);
		B[state] = ((num_t) k * (j + 1)) / NJ;
	});

	noarr::traverser(C).for_each([=](auto state) {
		auto [l, j] = noarr::get_indices<'l', 'j'>(state);
		C[state] = ((num_t) l * (j + 3)) / NL;
	});

	noarr::traverser(D).for_each([=](auto state) {
		auto [i, l] = noarr::get_indices<'i', 'l'>(state);
		D[state] = ((num_t) i * (l + 2)) / NK;
	});
}

template<class inner_t, class tmp_t, class A_t, class B_t>
__global__ void kernel_2mm_1(inner_t inner, num_t alpha, [[maybe_unused]] num_t beta, tmp_t tmp, A_t A, B_t B) {
	inner.template for_dims<'t', 's'>([=](auto inner) {
		auto state = inner.state();
		tmp[state] = 0;

		inner.template for_each<'k'>([=](auto state) {
			tmp[state] += alpha * A[state] * B[state];
		});
	});
}

template<class inner_t, class tmp_t, class C_t, class D_t>
__global__ void kernel_2mm_2(inner_t inner, [[maybe_unused]] num_t alpha, num_t beta, tmp_t tmp, C_t C, D_t D) {
	inner.template for_dims<'v', 's'>([=](auto inner) {
		auto state = inner.state();
		D[state] *= beta;

		inner.template for_each<'j'>([=](auto state) {
			D[state] += tmp[state] * C[state];
		});
	});
}

// run kernels
void run_2mm(num_t alpha, num_t beta, auto tmp, auto A, auto B, auto C, auto D) {
	// tmp: ni x nj
	// A: ni x nk
	// B: nk x nj
	// C: nl x nj
	// D: ni x nl

	auto trav1 = noarr::traverser(tmp, A, B)
		.order(noarr::into_blocks_dynamic<'i', 'I', 'i', 's'>(DIM_THREAD_BLOCK_Y))
		.order(noarr::into_blocks_dynamic<'j', 'J', 'j', 't'>(DIM_THREAD_BLOCK_X));
	
	auto trav2 = noarr::traverser(tmp, C, D)
		.order(noarr::into_blocks_dynamic<'i', 'I', 'i', 's'>(DIM_THREAD_BLOCK_Y))
		.order(noarr::into_blocks_dynamic<'l', 'L', 'l', 'v'>(DIM_THREAD_BLOCK_X));

	noarr::cuda_threads<'J', 'j', 'I', 'i'>(trav1)
		.simple_run(kernel_2mm_1, 0, alpha, beta, tmp, A, B);

	CUCH(hipGetLastError()); // check for configuration errors
	CUCH(hipDeviceSynchronize()); // join, check for execution errors
	
	noarr::cuda_threads<'L', 'l', 'I', 'i'>(trav2)
		.simple_run(kernel_2mm_2, 0, alpha, beta, tmp, C, D);

	CUCH(hipGetLastError()); // check for configuration errors
	CUCH(hipDeviceSynchronize()); // join, check for execution errors
}

class experiment : public virtual_experiment {
	template<class TMP, class A, class B, class C, class D>
	struct experiment_data : public virtual_data {
		TMP tmp;
		A a;
		B b;
		C c;
		D d;
		num_t alpha = 0;
		num_t beta = 0;

		experiment_data(TMP tmp, A a, B b, C c, D d)
			: tmp(std::move(tmp)), a(std::move(a)), b(std::move(b)), c(std::move(c)), d(std::move(d)) { }

		void run() override {
			run_2mm(alpha, beta, tmp.get_device_ref(), a.get_device_ref(), b.get_device_ref(), c.get_device_ref(), d.get_device_ref());
		}

		void print_results(std::ostream& os) override {
			d.fetch_to_host();
			noarr::serialize_data(os, d.get_host_ref() ^ noarr::hoist<'i'>());
		}
	};

public:
	experiment() {
		// problem size
		std::size_t ni = NI;
		std::size_t nj = NJ;
		std::size_t nk = NK;
		std::size_t nl = NL;

		cudaInit();

		// data
		experiment_data new_data{
			managed_bag(noarr::scalar<num_t>() ^ noarr::sized_vectors<'j', 'i'>(nj, ni)),
			managed_bag(noarr::scalar<num_t>() ^ noarr::sized_vectors<'k', 'i'>(nk, ni)),
			managed_bag(noarr::scalar<num_t>() ^ noarr::sized_vectors<'j', 'k'>(nj, nk)),
			managed_bag(noarr::scalar<num_t>() ^ noarr::sized_vectors<'j', 'l'>(nj, nl)),
			managed_bag(noarr::scalar<num_t>() ^ noarr::sized_vectors<'l', 'i'>(nl, ni))
		};

		init(new_data.alpha, new_data.beta, new_data.a.get_host_ref(), new_data.b.get_host_ref(), new_data.c.get_host_ref(), new_data.d.get_host_ref());

		new_data.a.fetch_to_device();
		new_data.b.fetch_to_device();
		new_data.c.fetch_to_device();
		new_data.d.fetch_to_device();

		data = std::make_unique<decltype(new_data)>(std::move(new_data));
	}
};


} // namespace

REGISTER_EXPERIMENT(2mm);
