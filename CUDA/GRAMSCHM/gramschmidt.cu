#include "hip/hip_runtime.h"
#include <memory>

#include <noarr/structures_extended.hpp>
#include <noarr/structures/extra/traverser.hpp>
#include <noarr/structures/interop/bag.hpp>
#include <noarr/structures/interop/serialize_data.hpp>
#include <noarr/structures/interop/cuda_traverser.cuh>

#include "common.hpp"
#include "defines.cuh"
#include "gramschmidt.cuh"
#include "noarr/structures/extra/shortcuts.hpp"

using num_t = DATA_TYPE;

namespace {

// initialize data
void init(auto A, auto R, auto Q) {
	// A: i x k
	// R: k x j
	// Q: i x k

	auto ni = A | noarr::get_length<'i'>();
	auto nj = R | noarr::get_length<'j'>();

	noarr::traverser(A, Q).for_each([=](auto state) {
		auto i = noarr::get_index<'i'>(state);
		auto k = noarr::get_index<'k'>(state);

		A[state] = ((num_t) i * k) / ni;
		Q[state] = ((num_t) i * (k + 1)) / nj;
	});

	noarr::traverser(R).for_each([=](auto state) {
		auto [k, j] = noarr::get_indices<'k', 'j'>(state);

		R[state] = ((num_t) k * (j + 2)) / nj;
	});
}

template<class inner_t, class A_t, class R_t, class Q_t>
__global__ void gramschmidt_kernel1(inner_t inner, A_t A, R_t R_diag, [[maybe_unused]] Q_t Q) {
	// A: i x k
	// R: k x j
	// Q: i x k

	inner.template for_dims<'t'>([=](auto inner) {
		auto state = inner.state();

		num_t nrm = 0;

		inner.template for_each<'i'>([=, &nrm](auto state) {
			nrm += A[state] * A[state];
		});

		R_diag[state] = sqrt(nrm);
	});
}

template<class inner_t, class A_t, class R_t, class Q_t>
__global__ void gramschmidt_kernel2(inner_t inner, A_t A, R_t R_diag, Q_t Q) {
	// A: i x k
	// R: k x j
	// Q: i x k

	inner.template for_each<'s'>([=](auto state) {
		Q[state] = A[state] / R_diag[state];
	});
}

template<class inner_t, class A_t, class R_t, class Q_t>
__global__ void gramschmidt_kernel3(inner_t inner, A_t A_ij, R_t R, Q_t Q) {
	// A: i x k
	// R: k x j
	// Q: i x k

	inner.template for_dims<'t'>([=](auto inner) {
		auto state = inner.state();
		auto [j, k] = noarr::get_indices<'j', 'k'>(state);

		if (j <= k)
			return;


		R[state] = 0;

		inner.template for_each<'i'>([=](auto state) {
			R[state] += Q[state] * A_ij[state];
		});

		inner.template for_each<'i'>([=](auto state) {
			A_ij[state] -= Q[state] * R[state];
		});
	});
}

// run kernels
void run_gramschmidt(auto A, auto R, auto Q) {
	// A: i x k
	// R: k x j
	// Q: i x k

	auto trav = noarr::traverser(A, R, Q);

	// `A_ij = A ^ noarr::rename<'k', 'j'>()` currently triggers a compiler bug, this is a simple workaround
	auto A_ij = noarr::make_bag(noarr::scalar<num_t>() ^ noarr::vectors_like<'j', 'i'>(trav.top_struct()), A.data());

	trav.template for_dims<'k'>([=](auto inner) {
		auto trav1 = inner
			.order(noarr::slice<'j'>(0, 1))
			.order(noarr::into_blocks_dynamic<'j', 'J', 'j', 't'>(DIM_THREAD_BLOCK_X))
			.order(noarr::bcast<'Y'>(1) ^ noarr::bcast<'y'>(DIM_THREAD_BLOCK_Y))
			;
	
		auto trav2 = inner
			.order(noarr::into_blocks_dynamic<'i', 'I', 'i', 's'>(DIM_THREAD_BLOCK_X))
			.order(noarr::bcast<'Y'>(1) ^ noarr::bcast<'y'>(DIM_THREAD_BLOCK_Y))
			;

		auto trav3 = inner
			.order(noarr::into_blocks_dynamic<'j', 'J', 'j', 't'>(DIM_THREAD_BLOCK_X))
			.order(noarr::bcast<'Y'>(1) ^ noarr::bcast<'y'>(DIM_THREAD_BLOCK_Y))
			;

		auto R_diag = R ^ noarr::fix<'j'>(noarr::get_index<'k'>(inner.state()));

		noarr::cuda_threads<'J', 'j', 'Y', 'y'>(trav1)
			.simple_run(gramschmidt_kernel1, 0, A, R_diag, Q);

		CUCH(hipGetLastError());
		CUCH(hipDeviceSynchronize());

		noarr::cuda_threads<'I', 'i', 'Y', 'y'>(trav2)
			.simple_run(gramschmidt_kernel2, 0, A, R_diag, Q);
		
		CUCH(hipGetLastError());
		CUCH(hipDeviceSynchronize());

		noarr::cuda_threads<'J', 'j', 'Y', 'y'>(trav3)
			.simple_run(gramschmidt_kernel3, 0, A_ij, R, Q);
		
		CUCH(hipGetLastError());
		CUCH(hipDeviceSynchronize());
	});
}

class experiment : public virtual_experiment {
	template<class A, class R, class Q>
	struct experiment_data : public virtual_data {
		A a;
		R r;
		Q q;

		experiment_data(A a, R r, Q q)
			: a(std::move(a)), r(std::move(r)), q(std::move(q)) { }

		void run() override {
			run_gramschmidt(a.get_device_ref(), r.get_device_ref(), q.get_device_ref());
		}

		void print_results(std::ostream& os) override {
			a.fetch_to_host();
			noarr::serialize_data(os, a.get_host_ref() ^ noarr::hoist<'i'>());
		}
	};

public:
	experiment() {
		// problem size
		std::size_t ni = NI;
		std::size_t nj = NJ;

		cudaInit();

		// data
		experiment_data new_data{
			managed_bag(noarr::scalar<num_t>() ^ noarr::sized_vectors<'k', 'i'>(nj, ni)),
			managed_bag(noarr::scalar<num_t>() ^ noarr::sized_vectors<'j', 'k'>(nj, nj)),
			managed_bag(noarr::scalar<num_t>() ^ noarr::sized_vectors<'k', 'i'>(nj, ni))
		};

		init(new_data.a.get_host_ref(), new_data.r.get_host_ref(), new_data.q.get_host_ref());

		new_data.a.fetch_to_device();
		new_data.r.fetch_to_device();
		new_data.q.fetch_to_device();

		data = std::make_unique<decltype(new_data)>(std::move(new_data));
	}
};


} // namespace

REGISTER_EXPERIMENT(gramschmidt);
